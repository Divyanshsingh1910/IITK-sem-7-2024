#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hwkernel(){
  printf("Hello world!\n");
}

int main(){
  //asynchronous function call 
  hwkernel<<<1,1>>>();
  hipDeviceSynchronize();
  return 0;
}
